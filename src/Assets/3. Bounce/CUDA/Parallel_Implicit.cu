#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "float3x3.hu"
#include "float3operators.hu"
#include "math.h"

#include "cstring_device.hu"
#include <cstdio>
#include "printf/printf.hu"

enum HyperelasticModelType
{
    StVK,
    neoHookean
} modelType;

float3 *Force, *d_X, *V;

int *d_Tet;
float3x3 *d_inv_Dm;
float *d_det_Dm;

float3 *V_sum;
int *V_num;

int number = 0, tet_number = 0;
float dt, s0, s1, damp, mass, floorY;

bool useGravity = true, laplacianSmoothing = true;

#define MAX_DEBUG_BUFFER_SIZE 20480
__managed__ int debug_tet_id = -1, debug_info_size = 0;
__managed__ char debug_info[MAX_DEBUG_BUFFER_SIZE];

// General update function, accepting a function pointer for the update function
template <typename T>
__global__ void verticesUpdate(float3 *arr, int number, T update)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= number)
        return;

    update(arr, i);
}

__device__ __forceinline__
    float3x3
    Build_Edge_Matrix(float3 *X, int *Tet, int tet)
{
    float3x3 ret;

    float3 X0 = X[Tet[tet * 4 + 0]];
    float3 X1 = X[Tet[tet * 4 + 1]];
    float3 X2 = X[Tet[tet * 4 + 2]];
    float3 X3 = X[Tet[tet * 4 + 3]];

    ret.setColumn(0, X0 - X1);
    ret.setColumn(1, X0 - X2);
    ret.setColumn(2, X0 - X3);

    return ret;
}

__global__ void _preUpdate(float3 *Force, int number, bool useGravity)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= number)
        return;

    Force[i] = make_float3(0, -9.81, 0) * useGravity;
}

__global__ void _preUpdate2(float3 *X, float3 *Force, int number, float mass, float dt)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= number)
        return;

    X[i] += Force[i] / mass / dt / dt / 2;
}

__device__ float2 __solve2x2_sym(float l1, float l2, float2 b)
{
    float detA = l1 * l1 - l2 * l2;
    return make_float2(
        (l1 * b.x - l2 * b.y) / detA,
        (l1 * b.y - l2 * b.x) / detA);
}

struct SVDResult
{
    float3x3 U, V;
    float3 L;
    __both__
    SVDResult() {}
    __both__
    SVDResult(float3x3 U, float3 L, float3x3 V) : U(U), L(L), V(V) {}
};

__device__ SVDResult _calcDGSVD(float3 *X, int *Tet, float3x3 *inv_Dm, int tet)
{
    float3x3 F = Build_Edge_Matrix(X, Tet, tet) * inv_Dm[tet];

    float3x3 U, V;
    float3 L;
    F.svd(U, L, V);

    return SVDResult(U, L, V);
}

__global__ void _calcForce_kernel(float3 *X, float3 *Force, int *Tet, float3x3 *inv_Dm, float *det_Dm, int tet_number, float s0, float s1)
{
    int tet = blockIdx.x * blockDim.x + threadIdx.x;
    if (tet >= tet_number)
        return;

    SVDResult svdres = _calcDGSVD(X, Tet, inv_Dm, tet);
    float3x3 &U = svdres.U, &V = svdres.V;
    float3 &L = svdres.L;

    float I = sqrMagnitude(L);
    float3 S = L * L;
    float3x3 dWdLd = float3x3(
        0.5f * L.x * (I - 3) * s0 + 2 * L.x * (S.x - 1) * s1,
        0.5f * L.y * (I - 3) * s0 + 2 * L.y * (S.y - 1) * s1,
        0.5f * L.z * (I - 3) * s0 + 2 * L.z * (S.z - 1) * s1);
    float3x3 P = U * dWdLd * V.transpose();
    float3x3 force = (-det_Dm[tet] / 6) * P * inv_Dm[tet].transpose();

    atomicAdd(&Force[Tet[tet * 4 + 0]], -force * make_float3(1, 1, 1));
    atomicAdd(&Force[Tet[tet * 4 + 1]], force.getColumn(0));
    atomicAdd(&Force[Tet[tet * 4 + 2]], force.getColumn(1));
    atomicAdd(&Force[Tet[tet * 4 + 3]], force.getColumn(2));

    /*if (debug_tet_id != -1 && tet == debug_tet_id && debug_info_size < MAX_DEBUG_BUFFER_SIZE)
    {
        char *tail = debug_info + strlen_d(debug_info);
        tail = strcat_d(tail, "Force: ");
        tail = to_string(force, tail);
        tail = strcat_d(tail, "\n");
        debug_tet_id = -1;
        debug_info_size = strlen_d(debug_info);
    }*/
}

void _calcForce()
{
    // Initiate gravity
    _preUpdate<<<(number + 255) / 256, 256>>>(Force, number, useGravity);
    hipDeviceSynchronize();

    // Calculate elastic forces
    _calcForce_kernel<<<(tet_number + 255) / 256, 256>>>(d_X, Force, d_Tet, d_inv_Dm, d_det_Dm, tet_number, s0, s1);
    hipDeviceSynchronize();
}

__global__ void _calcHessian(float3 *X, float3x3 *H, int *Tet, float3x3 *inv_Dm, float *det_Dm, int tet_number, float s0, float s1)
{
    int tet = blockIdx.x * blockDim.x + threadIdx.x;
    if (tet >= tet_number)
        return;

    SVDResult svdres = _calcDGSVD(X, Tet, inv_Dm, tet);
    float3x3 &U = svdres.U, &V = svdres.V;
    float3 &L = svdres.L;

    // Perturb L to avoid instabilities when lambdas are close to each other
    const float eps = 1e-6;
    if (L.x - L.y < eps)
        L.x += eps;
    if (L.y - L.z < eps)
        L.y += eps;
    if (L.x - L.y < eps)
        L.x += eps;

    float I = sqrMagnitude(L);
    float3 S = L * L;
    float3x3 dWdLd = float3x3(
        0.5f * L.x * (I - 3) * s0 + 2 * L.x * (S.x - 1) * s1,
        0.5f * L.y * (I - 3) * s0 + 2 * L.y * (S.y - 1) * s1,
        0.5f * L.z * (I - 3) * s0 + 2 * L.z * (S.z - 1) * s1);
    float3x3 P = U * dWdLd * V.transpose();

    float3x3 d2WdL2 = s0 * outer(L, L) + float3x3(0.5f * s0 * (I - 3)) + 2 * s1 * float3x3(3 * S - make_float3(1, 1, 1));
    float3x3 Ld = float3(L), dfmi_dFnl[3][3]; // Where the subscripts are n, l

    float3x3 debug0[3][3], debug1[3][3];

    for (int k = 0; k < 3; k++)
        for (int l = 0; l < 3; l++)
        {
            float3x3 dPdFkl, omegaU, omegaVt;

            // Calculate \frac{\partial\lambda_d}{\partial F_{kl}},
            // leading to central addend of \frac{\partial f}{\partial F_{kl}}
            float3x3 Ut_dFdFkl_V = outer(U.getRow(k), V.getRow(l));
            dPdFkl += float3x3(d2WdL2 * Ut_dFdFkl_V.diag());

            // Calculate U\frac{\partial U}{\partial F_{kl}} and \frac{\partial V^T}{\partial F_{kl}}V,
            // leading to the first and last addend of \frac{\partial f}{\partial F_{kl}}
            float2 U_Vt_01 = __solve2x2_sym(L.y, L.x, make_float2(Ut_dFdFkl_V.m01, -Ut_dFdFkl_V.m10));
            float2 U_Vt_02 = __solve2x2_sym(L.z, L.x, make_float2(Ut_dFdFkl_V.m02, -Ut_dFdFkl_V.m20));
            float2 U_Vt_12 = __solve2x2_sym(L.z, L.y, make_float2(Ut_dFdFkl_V.m12, -Ut_dFdFkl_V.m21));
            omegaU.m01 = U_Vt_01.x, omegaVt.m01 = U_Vt_01.y;
            omegaU.m02 = U_Vt_02.x, omegaVt.m02 = U_Vt_02.y;
            omegaU.m12 = U_Vt_12.x, omegaVt.m12 = U_Vt_12.y;
            omegaU += -omegaU.transpose(), omegaVt += -omegaVt.transpose();
            dPdFkl += omegaU * dWdLd + dWdLd * omegaVt;
            dPdFkl = U * dPdFkl * V.transpose();
            debug1[k][l] = dPdFkl;

            // Work out \frac{\partial f}{\partial F_{kl}} and add it to the total derivative
            dfmi_dFnl[k][l] = (-det_Dm[tet] / 6) * dPdFkl * inv_Dm[tet].transpose();
            debug0[k][l] = Ut_dFdFkl_V - omegaU * Ld - Ld * omegaVt;
        }

    // Work out Hessian = (\frac{\vec{f}_i}{\vec{x}_j})_{mn} = \frac{\partial f_{mi}}{\partial F_{nl}} * X^{-T}_{lj}
    // for all {i, j}s, where i, j are the indices of the vertices of the tetrahedron and m, n are the indices of Hessians
    // Which means we'd do 9 matrix multiplications to get 9 Hessian blocks.
    /*float3x3 Ht[3][3]; // Ht[i][j] = \frac{\vec{f}_i}{\vec{x}_j}
    for (int i = 0; i < 3; i++)
        for (int j = 0; j < 3; j++)
        {
            float3x3 H;
            for (int m = 0; m < 3; m++)
                for (int n = 0; n < 3; n++)
                {
                    H(m, n) = dfmi_dFnl[n][0](m, i) * inv_Dm[tet](j, 0)    //
                              + dfmi_dFnl[n][1](m, i) * inv_Dm[tet](j, 1)  //
                              + dfmi_dFnl[n][2](m, i) * inv_Dm[tet](j, 2); //
                }
            Ht[i][j] = H;
        }*/

    // To make it easy, we only take the diagonal blocks of the Hessian matrix
    float3x3 diagHt[4]; // Ht[i] = \frac{\vec{f}_i}{\vec{x}_i}

    // Notice: These are in fact i = 1, 2, 3. This for-loop definition is just for 0-based indexing.
    // We will compute i = 0 later due to its special form.
    for (int i = 0; i < 3; i++)
    {
        float3x3 diagH;
        for (int m = 0; m < 3; m++)
            for (int n = 0; n < 3; n++)
                diagH(m, n) = dfmi_dFnl[n][0](m, i) * inv_Dm[tet](i, 0)    //
                              + dfmi_dFnl[n][1](m, i) * inv_Dm[tet](i, 1)  //
                              + dfmi_dFnl[n][2](m, i) * inv_Dm[tet](i, 2); //

        atomicAdd(&H[Tet[tet * 4 + i + 1]], diagH); // 0-base to 1-base
        diagHt[i + 1] = diagH;
    }
    // Calculate \frac{\partial f_0}\frac{\partial x_0}, with the equations:
    // \frac{\partial f_0}{\partial x_0} = -\sum_{i=1}^{3} \frac{\partial f_i}{\partial x_0}
    // and d_0 = \frac{\partial F}{\partial x_0} = -(d_1 + d_2 + d_3)
    float3 d0 = inv_Dm[tet].getRow(0) + inv_Dm[tet].getRow(1) + inv_Dm[tet].getRow(2); // Two negations cancel out so we don't need to negate it here
                                                                                       // Also note that in the equation it is D_m^(-T) so d_i's are rows of D_m^(-1)

    for (int m = 0; m < 3; m++)
        for (int n = 0; n < 3; n++)
        {
            float h = 0;
            // Still i is in fact 1, 2, 3
            for (int i = 0; i < 3; i++)
                h += dfmi_dFnl[n][0](m, i) * at(d0, 0)    //
                     + dfmi_dFnl[n][1](m, i) * at(d0, 1)  //
                     + dfmi_dFnl[n][2](m, i) * at(d0, 2); //

            // Clamp h to force positive semi-definiteness
            /*if (h < 0)
                h = 0;*/
            diagHt[0](m, n) = h;
        }
    atomicAdd(&H[Tet[tet * 4]], diagHt[0]);

    if (debug_tet_id != -1 && tet == debug_tet_id && debug_info_size < MAX_DEBUG_BUFFER_SIZE)
    {
        char *tail = debug_info + strlen_d(debug_info);
        tail = strcat_d(tail, "Ht: \n");
        for (int i = 0; i < 4; i++)
        {
            tail = to_string(diagHt[i], tail);
            tail = strcat_d(tail, "\n");
        }
        /*tail = strcat_d(tail, "debug0: \n");
        for (int i = 0; i < 3; i++)
            for (int j = 0; j < 3; j++)
            {
                tail = to_string(debug0[i][j], tail);
                tail = strcat_d(tail, "\n");
            }
        tail = strcat_d(tail, "debug1: \n");
        for (int i = 0; i < 3; i++)
            for (int j = 0; j < 3; j++)
            {
                tail = to_string(debug1[i][j], tail);
                tail = strcat_d(tail, "\n");
            }*/
        // debug_tet_id = -1;
        debug_info_size = strlen_d(debug_info);
    }
}

__global__ void _newtonIterationStep(float3 *X, const float3 *X_hat, const float3 *Force, const float3x3 *H, int number, float s0, float s1, float mass, float dt)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= number)
        return;

    float K = mass / dt / dt;
    // float3 dX = (float3x3(K) + H[i]).inverse() * (-K * (X[i] - X_hat[i]) + Force[i]);

    float3 dX = (-K * (X[i] - X_hat[i]) + Force[i]) / (K * make_float3(1, 1, 1) + clampComponents(H[i].diag(), 0));
    // float mag = sqrMagnitude(dX);
    // if (isfinite(sqrMagnitude(Force[i])))
    {
        X[i] += dX;
    }
}

__global__ void _particleUpdate(
    float3 *X, float3 *V, float3 *Force, int number,
    float dt, float damp, float mass)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= number)
        return;

    // Explicit Euler
    // V[i] += Force[i] * dt / mass;
    V[i] *= damp;
    X[i] += V[i] * dt;
}

__global__ void _particleUpdatePost(
    float3 *X, float3 *last_X, float3 *V, int number,
    float dt, float damp, float mass, float floorY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= number)
        return;

    V[i] = (X[i] - last_X[i]) / dt;
    // V[i] *= damp;

    // Collision
    if (X[i].y < floorY)
    {
        V[i].y += (floorY - X[i].y) / dt;
        X[i].y = floorY;
    }
}

__global__ void _laplacianSmoothingTet(float3 *V, float3 *V_sum, int *V_num, int *Tet, int tet_number)
{
    int tet = blockIdx.x * blockDim.x + threadIdx.x;
    if (tet >= tet_number)
        return;

    float3 v = V[Tet[tet * 4 + 0]] + V[Tet[tet * 4 + 1]] + V[Tet[tet * 4 + 2]] + V[Tet[tet * 4 + 3]];
    atomicAdd(&V_sum[Tet[tet * 4 + 0]], v - V[Tet[tet * 4 + 0]]);
    atomicAdd(&V_sum[Tet[tet * 4 + 1]], v - V[Tet[tet * 4 + 1]]);
    atomicAdd(&V_sum[Tet[tet * 4 + 2]], v - V[Tet[tet * 4 + 2]]);
    atomicAdd(&V_sum[Tet[tet * 4 + 3]], v - V[Tet[tet * 4 + 3]]);
    atomicAdd(&V_num[Tet[tet * 4 + 0]], 1);
    atomicAdd(&V_num[Tet[tet * 4 + 1]], 1);
    atomicAdd(&V_num[Tet[tet * 4 + 2]], 1);
    atomicAdd(&V_num[Tet[tet * 4 + 3]], 1);
}

// Laplacian smoothing for vertices, omega is the relaxation factor (0.0 - 1.0, 0 for no smoothing)
__global__ void _laplacianSmoothingVert(float3 *V, float3 *V_sum, int *V_num, int number, float omega)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= number)
        return;

    V[i] = omega * V_sum[i] / (V_num[i] * 3) + (1 - omega) * V[i];
}

void _update(int iteration_number)
{
    int grid_size_vert = (number + 255) / 256;
    int grid_size_tet = (tet_number + 255) / 256;

    //_preUpdate<<<grid_size_vert, 256>>>(Force, number, useGravity);
    // hipDeviceSynchronize();

    float3x3 *H;
    float3 *last_X, *X_hat;
    hipMalloc(&H, number * sizeof(float3x3));
    hipMemset(H, 0, number * sizeof(float3x3));
    hipMalloc(&last_X, number * sizeof(float3));
    hipMalloc(&X_hat, number * sizeof(float3));

    // Do explicit integration for initial guess of X_hat
    hipMemcpy(last_X, d_X, number * sizeof(float3), hipMemcpyDeviceToDevice);
    _particleUpdate<<<grid_size_vert, 256>>>(d_X, V, Force, number, dt, damp, mass);
    hipDeviceSynchronize();
    hipMemcpy(X_hat, d_X, number * sizeof(float3), hipMemcpyDeviceToDevice);

    /*_calcForce();
    if (useGravity)
        _preUpdate2<<<grid_size_vert, 256>>>(d_X, Force, number, mass, dt);*/

    // Calculate Hessian only once
    _calcHessian<<<grid_size_tet, 256>>>(d_X, H, d_Tet, d_inv_Dm, d_det_Dm, tet_number, s0, s1);
    hipDeviceSynchronize();

    for (int i = 0; i < iteration_number; i++)
    {
        _calcForce(); // Is not a kernel itself and has synchronization inside already

        _newtonIterationStep<<<grid_size_vert, 256>>>(d_X, X_hat, Force, H, number, s0, s1, mass, dt);
        hipDeviceSynchronize();
    }
    // Output force to debug buffer
    if (debug_tet_id != -1 && debug_info_size < MAX_DEBUG_BUFFER_SIZE)
    {
        char *tail = debug_info + debug_info_size;
        strcat(tail, "Force: ");
        tail += strlen(tail);
        for (int i = 0; i < 4; i++)
        {
            float3 force;
            int vert_index_tet;
            hipMemcpy(&vert_index_tet, d_Tet + (4 * debug_tet_id + i), sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&force, Force + vert_index_tet, sizeof(float3), hipMemcpyDeviceToHost);
            tail = to_string(force, tail);
            strcat(tail, "\n");
            tail += strlen(tail);
        }
        debug_info_size = strlen(debug_info);
    }

    _particleUpdatePost<<<grid_size_vert, 256>>>(d_X, V, last_X, number, dt, damp, mass, floorY);
    hipDeviceSynchronize();

    if (laplacianSmoothing && V_sum && V_num)
    {
        hipMemset(V_sum, 0, number * sizeof(float3));
        hipMemset(V_num, 0, number * sizeof(int));
        _laplacianSmoothingTet<<<grid_size_tet, 256>>>(V, V_sum, V_num, d_Tet, tet_number);
        hipDeviceSynchronize();
        _laplacianSmoothingVert<<<grid_size_vert, 256>>>(V, V_sum, V_num, number, 0.67);
        hipDeviceSynchronize();
    }

    hipFree(H);
    hipFree(last_X);
    hipFree(X_hat);
}

extern "C"
{
    __export__ char *CUDA_device_name()
    {
        hipDeviceProp_t device;
        hipGetDeviceProperties(&device, 0);
        char *label = new char[256];
        sprintf(label, "(%s) -- %d, %d", device.name, number, tet_number);
        return label;
    }

    __export__ bool GetDebugInfo(char *info)
    {
        memcpy(info, debug_info, MAX_DEBUG_BUFFER_SIZE);
        memset(debug_info, 0, MAX_DEBUG_BUFFER_SIZE);
        debug_info_size = 0;
        return true;
    }

    __export__ void SetDebugTet(int tet_id)
    {
        debug_tet_id = tet_id;
    }

    __export__ void Initialize(
        int *Tet, float3x3 *inv_Dm, float *det_Dm,
        int number, int tet_number, bool useGravity, bool laplacianSmoothing,
        float dt, float s0, float s1, float damp, float mass, float floorY,
        HyperelasticModelType modelType)
    {
        hipSetDeviceFlags(hipDeviceScheduleBlockingSync);

        // if (number != ::number)
        {
            if (::number)
            {
                hipFree(Force);
                hipFree(V);
                hipFree(V_sum);
                hipFree(V_num);
                hipFree(d_X);
            }
            ::number = number;
            hipMalloc(&V, number * sizeof(float3));
            hipMalloc(&Force, number * sizeof(float3));
            hipMalloc(&V_sum, number * sizeof(float3));
            hipMalloc(&V_num, number * sizeof(int));
            hipMalloc(&d_X, number * sizeof(float3));
        }

        // if (tet_number != ::tet_number)
        {
            if (::tet_number)
            {
                hipFree(d_Tet);
                hipFree(d_inv_Dm);
                hipFree(d_det_Dm);
            }
            ::tet_number = tet_number;
            hipMalloc(&d_Tet, tet_number * 4 * sizeof(int));
            hipMalloc(&d_inv_Dm, tet_number * sizeof(float3x3));
            hipMalloc(&d_det_Dm, tet_number * sizeof(float));
        }

        hipMemset(V, 0, number * sizeof(float3));
        hipMemset(Force, 0, number * sizeof(float3));
        hipMemset(V_sum, 0, number * sizeof(float3));
        hipMemset(V_num, 0, number * sizeof(int));

        hipMemcpy(d_Tet, Tet, tet_number * 4 * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_inv_Dm, inv_Dm, tet_number * sizeof(float3x3), hipMemcpyHostToDevice);
        hipMemcpy(d_det_Dm, det_Dm, tet_number * sizeof(float), hipMemcpyHostToDevice);

        ::dt = dt;
        ::s0 = s0;
        ::s1 = s1;
        ::damp = damp;
        ::mass = mass;
        ::floorY = floorY;
        ::useGravity = useGravity;
        ::laplacianSmoothing = laplacianSmoothing;
    }

    __export__ void Update(float3 *X, int iteration_number)
    {
        hipMemcpy(d_X, X, number * sizeof(float3), hipMemcpyHostToDevice);
        _update(iteration_number);
        hipMemcpy(X, d_X, number * sizeof(float3), hipMemcpyDeviceToHost);
    }

    __export__ void Impulse(float3 impulse)
    {
        verticesUpdate<<<(number + 255) / 256, 256>>>(
            V, number,
            [=] __device__(float3 * V, int i)
            { V[i] += impulse; });
        hipDeviceSynchronize();
    }
}